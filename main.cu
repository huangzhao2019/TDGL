#include "hip/hip_runtime.h"
#include "cuTDGL.h"
#include "hip/hip_complex.h"
#include <iostream>
#include "math.h"

__global__ void CUDAinitMesh();
__global__ void CUDAcalW(double *d_Ba);
__global__ void CUDAcalBC();
__global__ void CUDAcaldPsidt();
__global__ void CUDAcaldUdt();
__global__ void CUDAonestep();
__global__ void CUDACalTotal(double *d_Ba);
__global__ void TDGLoutput(hipDoubleComplex *dev_psi,double *dev_Jsx,double *dev_Jsy,double *dev_Ax,\
						   hipDoubleComplex *dev_Ux,double *dev_VOR,double *dev_ENG,double *dev_Bz);

int main(){
	hipDoubleComplex *dev_Psi;
	double *dev_Jsx;
	double *dev_Jsy;
	double *dev_Ax;
	hipDoubleComplex *dev_Ux;
	double *dev_VOR;
	double *dev_ENG;
	double *d_Ba;
	double *dev_Bz;
	width=Nx+1;
	height=Ny+1;
	sizeComplex=width*height*sizeof(hipDoubleComplex);
	sizeDouble=width*height*sizeof(double);
	hipMalloc(&dev_Psi,sizeComplex);
	hipMalloc(&dev_Jsx,sizeDouble);
	hipMalloc(&dev_Jsy,sizeDouble);
	hipMalloc(&dev_Ux,sizeComplex);
	hipMalloc(&d_Ba,sizeof(double));
	hipMalloc(&dev_VOR,sizeDouble);
	hipMalloc(&dev_ENG,sizeDouble);
	hipMalloc(&dev_Bz,sizeDouble);
	hipMalloc(&dev_Ax,sizeDouble);
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid((width+dimBlock.x-1)/dimBlock.x,(height+dimBlock.y-1)/dimBlock.y);
	CUDAinitMesh<<<dimGrid,dimBlock>>>();//initialize the mesh
	hipDeviceSynchronize();
    
	int sample=100;
	int i,prepare=96700,fileid=0,total=50,n;
	prepare*=sample;
	total*=sample;
	double sBa=10.5,eBa=20.3,stepBa1=0.01,stepBa2=-0.5,nowBa=4.75,interBa=100;
	nowBa=sBa;
	double sumMag=0.0,sumNumVor=0.0,sumSysEng=0.0;
	FILE *fpMag=fopen("Mag2.dat","w");
	fprintf(fpMag,"Ba\tMag\tNumVor\tSysEng\n");
	fclose(fpMag);
	while(nowBa<eBa&&nowBa>=0.0){
		n=0;
        hipMemcpy(d_Ba,&nowBa,sizeof(double),hipMemcpyHostToDevice);
		double magtest[300];
	    for(i=0;i<prepare;i++){
	        CUDAcalBC<<<dimGrid,dimBlock>>>();
//	        hipDeviceSynchronize();
	        CUDAcalW<<<dimGrid,dimBlock>>>(d_Ba);
//	        hipDeviceSynchronize();
	        CUDAcaldPsidt<<<dimGrid,dimBlock>>>();
//	        hipDeviceSynchronize();
			CUDAcaldUdt<<<dimGrid,dimBlock>>>();
//			hipDeviceSynchronize();
	        CUDAonestep<<<dimGrid,dimBlock>>>();
//	        hipDeviceSynchronize();
//		}
		if(i%(300*sample)==0){
                CUDACalTotal<<<dimGrid,dimBlock>>>(d_Ba);
	            hipDeviceSynchronize();
			    TDGLoutput<<<dimGrid,dimBlock>>>(dev_Psi,dev_Jsx,dev_Jsy,dev_Ax,dev_Ux,dev_VOR,dev_ENG,dev_Bz);
				hipDeviceSynchronize();
	            hipMemcpy(HGL.Psi,dev_Psi,sizeComplex,hipMemcpyDeviceToHost);
	            hipMemcpy(HGL.Jsx,dev_Jsx,sizeDouble,hipMemcpyDeviceToHost);
	            hipMemcpy(HGL.Jsy,dev_Jsy,sizeDouble,hipMemcpyDeviceToHost);
	            hipMemcpy(HGL.Ux,dev_Ux,sizeComplex,hipMemcpyDeviceToHost);
				hipMemcpy(HGL.VOR,dev_VOR,sizeDouble,hipMemcpyDeviceToHost);
				hipMemcpy(HGL.ENG,dev_ENG,sizeDouble,hipMemcpyDeviceToHost);
				hipMemcpy(HGL.Bz,dev_Bz,sizeDouble,hipMemcpyDeviceToHost);
				hipMemcpy(HGL.Ax,dev_Ax,sizeDouble,hipMemcpyDeviceToHost);

	            fileid++; 

				HGL.CalTotal(nowBa);
	            printf("%lf\t%lf\t%8.8lf\t%lf\n", HGL.magnetization,HGL.NumOfVortices,HGL.SysEng,nowBa);
				magtest[n]=HGL.magnetization;
				HGL.nameBa=nowBa;//add Ba to the title of file's name to find the corresponding file more conveniently
	            HGL.OutputPsi(fileid);
	            HGL.OutputBz(fileid);
	            HGL.OutputJsy(fileid);
	            HGL.OutputJsx(fileid);
				HGL.OutputPsiPhase(fileid);
				HGL.OutputAx(fileid);
				HGL.OutputUx(fileid);
				int N=300;
				if(nowBa<12.15)
					N=100;
				if(n>N&&abs(magtest[n]-magtest[n-1])<0.000001&&abs(magtest[n]-magtest[n-20])<0.000001)//get out of the loop when equilibrium arrives
					break;
				n++;
			}
		}
		for(i=0;i<total;i++){
	        CUDAcalBC<<<dimGrid,dimBlock>>>();
	        //hipDeviceSynchronize();
	        CUDAcalW<<<dimGrid,dimBlock>>>(d_Ba);
	        //hipDeviceSynchronize();
	        CUDAcaldPsidt<<<dimGrid,dimBlock>>>();
	        //hipDeviceSynchronize();
			CUDAcaldUdt<<<dimGrid,dimBlock>>>();
			//hipDeviceSynchronize();
	        CUDAonestep<<<dimGrid,dimBlock>>>();
	        //hipDeviceSynchronize();
            if(i%sample==0){
                CUDACalTotal<<<dimGrid,dimBlock>>>(d_Ba);
	            hipDeviceSynchronize();
			    TDGLoutput<<<dimGrid,dimBlock>>>(dev_Psi,dev_Jsx,dev_Jsy,dev_Ax,dev_Ux,dev_VOR,dev_ENG,dev_Bz);
				hipDeviceSynchronize();
	            hipMemcpy(HGL.Psi,dev_Psi,sizeComplex,hipMemcpyDeviceToHost);
//	            hipMemcpy(HGL.Jsx,dev_Jsx,sizeComplex,hipMemcpyDeviceToHost);
//	            hipMemcpy(HGL.Jsy,dev_Jsy,sizeComplex,hipMemcpyDeviceToHost);
//	            hipMemcpy(HGL.W,dev_W,sizeComplex,hipMemcpyDeviceToHost);
				hipMemcpy(HGL.VOR,dev_VOR,sizeDouble,hipMemcpyDeviceToHost);
				hipMemcpy(HGL.ENG,dev_ENG,sizeDouble,hipMemcpyDeviceToHost);
				hipMemcpy(HGL.Bz,dev_Bz,sizeDouble,hipMemcpyDeviceToHost);
				HGL.CalTotal(nowBa);
				sumMag+=HGL.magnetization;
				sumNumVor+=HGL.NumOfVortices;
				sumSysEng+=HGL.SysEng;
			}
		}
		sumSysEng=sumSysEng/(total/sample);
		sumMag=sumMag/(total/sample);
		sumNumVor=sumNumVor/(total/sample);
//		if(abs(sumMag)<0.0007)
//			interBa=nowBa-0.05;
		fpMag=fopen("Mag2.dat","a");
		fprintf(fpMag,"%lf\t%lf\t%lf\t%lf\n",nowBa,sumMag,sumNumVor,sumSysEng);
		fclose(fpMag);
		if(nowBa<interBa)
		    nowBa+=stepBa2;
		else
			nowBa+=stepBa1;
		sumSysEng=0.0;
        sumMag=0.0;
		sumNumVor=0.0;//change sumNumVor back to zero
	}
	hipFree(dev_Psi);
	hipFree(dev_Jsx);
	hipFree(dev_Jsy);
	hipFree(dev_Ux);
	hipFree(dev_VOR);
	hipFree(dev_ENG);
	hipFree(dev_Bz);
	hipFree(dev_Ax);
	return 1;
}


__global__ void CUDAinitMesh(){//initialize the parameters in mesh
    int row=blockIdx.y*blockDim.y+threadIdx.y;
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	if(row<Ny+1&&col<Nx+1)
        CGL.CUinitMesh(row,col);
}

__global__ void CUDAcalBC(){//calculate the boundary condition
    int row=blockIdx.y*blockDim.y+threadIdx.y;
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	if(row<Ny+1&&col<Nx+1)
	    CGL.CUcalBC(row,col);
}

__global__ void CUDAcalW(double *d_Ba){//calculate W which will be used to calculate dUi/dt
    int row=blockIdx.y*blockDim.y+threadIdx.y;
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	if(row<Ny+1&&col<Nx+1)
	    CGL.CUcalW(row,col,d_Ba);
}

__global__ void CUDAcaldPsidt(){//calculate dUi/dt and dPsi/dt
    int row=blockIdx.y*blockDim.y+threadIdx.y;
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	if(row<Ny+1&&col<Nx+1)
	    CGL.CUcaldPsidt(row,col);
}

__global__ void CUDAcaldUdt(){
    int row=blockIdx.y*blockDim.y+threadIdx.y;
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	if(row<Ny+1&&col<Nx+1)
        CGL.CUcaldUdt(row,col);
}

__global__ void CUDAonestep(){//onestep forward
    int row=blockIdx.y*blockDim.y+threadIdx.y;
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	if(row<Ny+1&&col<Nx+1)
	    CGL.CUonestep(row,col);
}

__global__ void CUDACalTotal(double *d_Ba){//calculate the number of vortices and system energy per point
    int row=blockIdx.y*blockDim.y+threadIdx.y;
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	if(row<Ny+1&&col<Nx+1){
	    CGL.CUcalNumOfVortices(row,col);
	    CGL.CUcalSysEng(row,col,d_Ba);
//		CGL.CUcalBC(row,col);
		CGL.CUcalJs(row,col);
		CGL.CUcaldPsidt(row,col);//need to calculate the boundary current to cover the Js calculated just now
	}
}


__global__ void TDGLoutput(hipDoubleComplex *dev_Psi,double *dev_Jsx,double *dev_Jsy,double *dev_Ax,\
						   hipDoubleComplex *dev_Ux,double *dev_VOR,double *dev_ENG,double *dev_Bz){//output the datas you choose
    int row=blockIdx.y*blockDim.y+threadIdx.y;
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	int d_width=Ny+1;
	if(row<Ny+1&&col<Nx+1){
	    dev_Psi[col*d_width+row]=CGL.d_Psi[col][row];
	    dev_Jsx[col*d_width+row]=CGL.d_Jsx[col][row];
	    dev_Jsy[col*d_width+row]=CGL.d_Jsy[col][row];
	    dev_Ux[col*d_width+row]=CGL.d_Ux[col][row];
	    dev_VOR[col*d_width+row]=CGL.d_VOR[col][row];
	    dev_ENG[col*d_width+row]=CGL.d_ENG[col][row];
	    dev_Bz[col*d_width+row]=CGL.d_Bz[col][row];
		dev_Ax[col*d_width+row]=CGL.d_Ax[col][row];
	}
}
